/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-1
 * Description: Computation of a matrix C = Kronecker_prod(A, B.T)
 *              where A and B are matrices of dimension (m, n) and
 *              the output is of the dimension (m * n, m * n). 
 * Note: All lines marked in --> should be replaced with code. 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void per_row_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
    //Taking Rows of A and B in thread and evaluating
    int rowA = blockIdx.x;
    int rowB = threadIdx.x;
    //To Map the kronocker multiplication in result using formulae
    for(int colA=0;colA<n;colA++)
    {
        for(int colB=0;colB<n;colB++)
        {
		int a = (rowA*n + colB)*m*n;
		int b = (rowB+m*colA);
		int c = rowA*n + colA;
		int d = rowB*n + colB;
            C[a + b] = A[c] * B[d];
        }
    }
}

__global__ void per_column_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
    //Taking column of matrix A using grid, depends on the column size and column of B directly from n
    int colA = blockIdx.x * blockDim.x + threadIdx.x;
    int colB = threadIdx.y;
    if(colA < n ) //checking out of bound condition
    { 
      for(int rowA = 0;rowA < m;rowA++)
      {
        for(int rowB = 0; rowB < m;rowB++)
          {
            int a = (rowA*n + colB)*m*n;
		int b = (rowB+m*colA);
		int c = rowA*n + colA;
		int d = rowB*n + colB;
                C[a + b] = A[c] * B[d]; //forumlae to map the multiplication as required
          }
      }
    }
}

__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
	//Calculating thread id  so that it will take id properly...
	int a = threadIdx.x*blockDim.y;
	int b = blockIdx.y*blockDim.y*blockDim.x;
	int c = blockIdx.x*gridDim.y*blockDim.y*blockDim.x;
	int d = threadIdx.y;
	int id = a + b + c + d;
    if(id<((m*n)*(m*n))) //checking for out of bound
    {
	int e = (id/(n*m*n));
	int f = (id/m)%n;
	int g = ((id/(m*n))%n);
        C[id] = A[e*n + f]*B[(id%m)*n + g]; // calculating the kronockers multiplication
	
    }
}

/**
 * Prints any 1D array in the form of a matrix
 **/
void printMatrix(long int *arr, long int rows, long int cols, char* filename){
    outfile.open(filename);
    for(long int i = 0; i < rows; i++){
        for(long int j = 0; j < cols; j++){
            outfile<<arr[i * cols + j]<<" ";
        }
        outfile<<"\n";
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    long int m,n;	
    cin>>m>>n;	

    // Host_arrays 
    long int *h_a,*h_b,*h_c;

    // Device arrays 
    long int *d_a,*d_b,*d_c;
	
    // Allocating space for the host_arrays 
    h_a = (long int *) malloc(m * n * sizeof(long int));
    h_b = (long int *) malloc(m * n * sizeof(long int));	
    h_c = (long int *) malloc(m * m * n * n * sizeof(long int));	

    // Allocating memory for the device arrays 
    // --> Allocate memory for A on device 
      hipMalloc(&d_a,sizeof(long int) * m * n);
    // --> Allocate memory for B on device 
      hipMalloc(&d_b,sizeof(long int) * m * n);
    // --> Allocate memory for C on device 
      hipMalloc(&d_c,sizeof(long int) * m * n * m * n);
    // Read the input matrix A 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_a[i];
    }

    //Read the input matrix B 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_b[i];
    }

    // Transfer the input host arrays to the device 
    // --> Copy A from Host to Device
    hipMemcpy(d_a,h_a,sizeof(long int) * m * n,hipMemcpyHostToDevice);
    // --> Copy B from Host to Device 
    hipMemcpy(d_b,h_b,sizeof(long int) * m * n,hipMemcpyHostToDevice);
    long int gridDimx, gridDimy;
    
    // Launch the kernels
    /**
     * Kernel 1 - per_row_AB_kernel
     * To be launched with 1D grid, 1D block
     * Each thread should process a complete row of A, B
     **/

    // --> Set the launch configuration
    dim3 grid1(m,1,1);
    dim3 block1(m,1,1); 


    double starttime = rtclock();  

    // --> Launch the kernel 
    per_row_AB_kernel<<<grid1,block1>>>(d_a,d_b,d_c,m,n);
    hipDeviceSynchronize();                                                           

    double endtime = rtclock(); 
	printtime("GPU Kernel-1 time: ", starttime, endtime);  

    // --> Copy C from Device to Host 
    hipMemcpy(h_c,d_c,sizeof(long int) * m * n * m * n,hipMemcpyDeviceToHost);
    

    printMatrix(h_c, m * n, m * n,"kernel1.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(int));

    /**
     * Kernel 2 - per_column_AB_kernel
     * To be launched with 1D grid, 2D block
     * Each thread should process a complete column of  A, B
     **/
    
    // --> Set the launch configuration 
    int gridDimx1 = ceil(float(n)/(10));
    dim3 grid2(gridDimx1,1,1);
    dim3 block2(10,n,1);

    starttime = rtclock(); 

    // --> Launch the kernel
    per_column_AB_kernel<<<grid2,block2>>>(d_a,d_b,d_c,m,n); 

    hipDeviceSynchronize(); 

    endtime = rtclock(); 
  	printtime("GPU Kernel-2 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
    hipMemcpy(h_c,d_c,sizeof(long int) * m * n * m * n,hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel2.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(int));

    /**
     * Kernel 3 - per_element_kernel
     * To be launched with 2D grid, 2D block
     * Each thread should process one element of the output 
     **/
    gridDimx = ceil(float(n * n) / 16);
    gridDimy = ceil(float(m * m) / 64);
    dim3 grid3(gridDimx,gridDimy,1);
    dim3 block3(64,16,1);

    starttime = rtclock();  

    // --> Launch the kernel 
    per_element_kernel<<<grid3,block3>>>(d_a,d_b,d_c,m,n);
    hipDeviceSynchronize();                                                              

    endtime = rtclock();  
	printtime("GPU Kernel-3 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
    hipMemcpy(h_c,d_c,sizeof(long int) * m * n * m * n,hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel3.txt");

    return 0;
}
